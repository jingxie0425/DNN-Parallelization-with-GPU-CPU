#include "hip/hip_runtime.h"
/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "cnn.h"
#include "cnncu.h"
#include "funcs.h"
#include "dense.h"

//call:
//grid = {oN}
//block = {oD,oD}
__global__
void relu(unsigned int oD,float* outputs){
	unsigned int oN = blockIdx.x;
	unsigned int oDX = threadIdx.x;
	unsigned int oDY = threadIdx.y;

	if(*(outputs+(oN)*oD*oD+oDY*oD+oDX) < 0){
		*(outputs+(oN)*oD*oD+oDY*oD+oDX) = 0;
	}

	__syncthreads();
}

__global__
void reluDense(float* out){
	unsigned int n = blockIdx.x;

	if(*(out+n) < 0){
		*(out+n) = 0;
	}

	__syncthreads();
}

