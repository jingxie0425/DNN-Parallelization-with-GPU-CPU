#include "hip/hip_runtime.h"
/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "dense.h"
#include "densecu.h"
#include "cnn.h"

DENSEHANDLER denseNew(unsigned int inputN,unsigned int preNodeNum, unsigned int nodeNum){
	//first allocate host side
	DENSEHANDLER densehandler;
	densehandler = (DENSEHANDLER)malloc(sizeof(DENSESTATE));//malloc cpu handler

	hipMalloc(&(densehandler->Weights),preNodeNum*nodeNum*sizeof(float));
	hipMalloc(&(densehandler->bias),nodeNum*sizeof(float));
	hipMalloc(&(densehandler->outs),inputN*nodeNum*sizeof(float));
	return densehandler;
}

HEADDENSEHANDLER headDenseNew(unsigned int inputN,unsigned int preNodeNum, unsigned int nodeNum){
	//first allocate host side
	HEADDENSEHANDLER headdensehandler;
	headdensehandler = (HEADDENSEHANDLER)malloc(sizeof(HEADDENSESTATE));//malloc cpu handler

	hipMalloc(&(headdensehandler->Weights),preNodeNum*nodeNum*sizeof(float));
	hipMalloc(&(headdensehandler->bias),nodeNum*sizeof(float));
	hipMalloc(&(headdensehandler->outs),inputN*nodeNum*sizeof(float));
	hipMalloc(&(headdensehandler->inputs),inputN*preNodeNum*sizeof(float));

	return headdensehandler;
}

void denseLoad(DENSEHANDLER densehandler,unsigned int preNodeNum, unsigned int nodeNum, float* w, float* b){
	hipMemcpy(densehandler->Weights,w,preNodeNum*nodeNum*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(densehandler->bias,b,nodeNum*sizeof(float),hipMemcpyHostToDevice);
}

void headDenseLoad(HEADDENSEHANDLER headdensehandler,unsigned int inputN,unsigned int preNodeNum, unsigned int nodeNum, float* w, float* b,float* in){
	hipMemcpy(headdensehandler->Weights,w,preNodeNum*nodeNum*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(headdensehandler->bias,b,nodeNum*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(headdensehandler->inputs,in,inputN*preNodeNum*sizeof(float),hipMemcpyHostToDevice);
}

/**
 * This function is for dense layer at edge of convolutional and dense (it includes flatten inside)
 * call
 * grid:{inN,oD*oD,fN} here oD is new oD after max pooling
 * block:{nodeN} 1024 maximum node number
 */
__global__
void forward(float* denseouts, float* denseweights, float* preOut,unsigned int preOD,unsigned int fN, unsigned int nodeNum){
	unsigned int inputIndex = blockIdx.x;
	unsigned int columnIndex = blockIdx.y;
	unsigned int rowIndex = blockIdx.z;
	unsigned int nodeIndex = threadIdx.x;
	float tmpMulti = *((preOut + inputIndex*preOD*preOD*fN) + (rowIndex*preOD*preOD + columnIndex)) * *(denseweights + (rowIndex + columnIndex*fN)*nodeNum + nodeIndex);
	atomicAdd(denseouts + inputIndex*nodeNum + nodeIndex,tmpMulti);
	__syncthreads();
}

/**
 * This function is for dense layer and dense layer
 * call
 * grid:{inN,preNodeNum,nodeN}
 * block:{1}
 */
__global__
void forwardD(float* predenseouts,float* currdensewights,float* currdenseouts, unsigned int preNodeNum, unsigned int nodeNum){
	unsigned int inputIndex = blockIdx.x;
	unsigned int preNodeIndex = blockIdx.y;
	unsigned int nodeIndex = blockIdx.z;
	float tmpMulti = *((predenseouts + inputIndex*preNodeNum) + preNodeIndex) * *(currdensewights + preNodeIndex*nodeNum + nodeIndex);
	atomicAdd(currdenseouts + inputIndex*nodeNum + nodeIndex,tmpMulti);
	__syncthreads();
}

/**
 * call
 * grid: {inN,nodeN}
 * block: {}
 */
__global__
void addBias(float* denseouts,float* densebias,unsigned int nodeNum){
	unsigned int inputIndex = blockIdx.x;
	unsigned int nodeIndex = blockIdx.y;
	*(denseouts + inputIndex*nodeNum + nodeIndex) = *(denseouts + inputIndex*nodeNum + nodeIndex) + *(densebias + nodeIndex);
	__syncthreads();
}
