/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include <iostream>
#include "cnn.h"
#include "cnncu.h"
#include "funcs.h"
#include "maxpooling.h"
#include "maxpool.h"

#define TIME

MAXPOOLHANDLER maxpoolNew(unsigned int steps, unsigned int oN, unsigned int oD){
	unsigned int newOD = oD/steps;
	MAXPOOLHANDLER maxpoolhandler = (MAXPOOLHANDLER)malloc(sizeof(MAXPOOLSTATE));
	hipMalloc(&(maxpoolhandler->maxpoolOuts),oN*newOD*newOD*sizeof(float));//malloc new space for maxpooling out

	return maxpoolhandler;
}

void maxpool(float* preOut,MAXPOOLHANDLER maxpoolhandler,unsigned int steps, unsigned int oN, unsigned int oD){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
	unsigned int newOD = oD/steps;
	maxpooling<<<oN,{newOD,newOD}>>>(preOut,oD,steps,maxpoolhandler->maxpoolOuts,newOD);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("maxpool takes %.4f ms\n", elapsed);
#endif
}

void maxpoolFree(MAXPOOLHANDLER maxpoolhandler){
	hipFree(maxpoolhandler->maxpoolOuts);
	free(maxpoolhandler);
}