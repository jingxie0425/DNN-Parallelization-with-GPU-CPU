#include "hip/hip_runtime.h"
/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "cnn.h"
#include "cnncu.h"

//#define DEBUG

CNNHANDLER CnnNew(unsigned int inD, unsigned int inN, unsigned int fD, unsigned int fN, unsigned int oD, unsigned int oN,unsigned int picNum){
	//for bias
	CNNHANDLER cnnhandler = (CNNHANDLER)malloc(sizeof(CNNSTATE));
	hipMalloc(&(cnnhandler->bias),fN*sizeof(float));
	//for filterweights
	hipMalloc(&(cnnhandler->filterWeights),fN*picNum*fD*fD*sizeof(float));
	//for inputs
	hipMalloc(&(cnnhandler->inputs),inN*picNum*inD*inD*sizeof(float));
	//for outputs
	hipMalloc(&(cnnhandler->outputs),oN*oD*oD*sizeof(float));

	return cnnhandler;
}



void CnnLoad(CNNHANDLER cnnhandler,unsigned int inD, unsigned int inN, unsigned int fD, unsigned int fN,unsigned int picNum, float* cpubias, float* cpufilterWeights, float* cpuinputs){
	//for bias
	hipMemcpy(cnnhandler->bias, cpubias,fN*sizeof(float),hipMemcpyHostToDevice);
	//for filterweights
	hipMemcpy(cnnhandler->filterWeights, cpufilterWeights,fN*picNum*fD*fD*sizeof(float),hipMemcpyHostToDevice);
	//for inputs
	hipMemcpy(cnnhandler->inputs, cpuinputs,inN*picNum*inD*inD*sizeof(float),hipMemcpyHostToDevice);
}

void CnnLoadnonHead(CNNHANDLER cnnhandler, unsigned int fD, unsigned int fN,unsigned int picNum, float* cpubias, float* cpufilterWeights){
	//for bias
	hipMemcpy(cnnhandler->bias, cpubias,fN*sizeof(float),hipMemcpyHostToDevice);
	//for filterweights
	hipMemcpy(cnnhandler->filterWeights, cpufilterWeights,fN*picNum*fD*fD*sizeof(float),hipMemcpyHostToDevice);
	//for inputs
	//hipMemcpy(cnnhandler->inputs, cpuinputs,inN*inD*inD*sizeof(float),hipMemcpyHostToDevice);
}

//call:
//grid = {inN,fN}
//block = {oD,oD}
__global__
void addBias(unsigned int oD,unsigned int fN,float* biasAry,float* outputs){
	unsigned int filterIdx = blockIdx.y;
	unsigned int oDX = threadIdx.x;
	unsigned int oDY = threadIdx.y;
	unsigned int inputIdx = blockIdx.x;

	float bias = *(biasAry+filterIdx);
	*(outputs+(inputIdx*fN+filterIdx)*oD*oD+oDY*oD+oDX) += bias;
	__syncthreads();
}

/*call:
 * grid:{inN,fN}
 * block:{1}
 * */
__global__
void inferencePicPickHead(float* inputs, float* filterWeights,float* outputs, unsigned int oD,unsigned int fD,unsigned int fN,unsigned int inD){
	unsigned int filterIdx = blockIdx.y;
	unsigned int inputIdx = blockIdx.x;

	float* iPic;
	float* oPic;
	float* fPic;

	iPic = inputs+inputIdx*inD*inD;
	fPic = filterWeights+filterIdx*fD*fD;
	oPic = outputs+(inputIdx*fN+filterIdx)*oD*oD;

	inferencePicLev<<<{oD,oD},{fD,fD}>>>(iPic, oPic, fPic, inD, fD, oD);
}

/*call:
 * inN: input sample amount
 * picN: pic amount of each sample
 * fN: filter amount, each filter has picN filterPic for pics in each sample
 * grid:{inN,picN,fN}
 * block:{1}
 * */
__global__
void inferencePicPickNonHead(float* inputs, float* filterWeights,float* outputs, unsigned int oD,unsigned int fD,unsigned int fN,unsigned int picN,unsigned int inD){
	unsigned int inIdx = blockIdx.x;
	unsigned int picIdx = blockIdx.y;
	unsigned int fIdx = blockIdx.z;

	float* iPic;
	float* oPic;
	float* fPic;

	//Insize = picN*inPicSize 	 	inPicSize = inD*inD
	iPic = inputs + inIdx*picN*inD*inD + picIdx*inD*inD;
	//filterPicSize = fD*fD
	fPic = filterWeights + picIdx*fN*fD*fD + fIdx*fD*fD;
	//outsize = oD*oD
	oPic = outputs + oD*oD*(inIdx*fN+fIdx);

	inferencePicLev<<<{oD,oD},{fD,fD}>>>(iPic, oPic, fPic, inD, fD, oD);
}

/*
 * call: grid{oD,oD}
 * 		 block{fD,fD}
 * */
__global__
void inferencePicLev(float* iPic, float* oPic, float* fPic, unsigned int inD, unsigned int fD, unsigned int oD){
	unsigned int tIDx = threadIdx.x;
	unsigned int tIDy = threadIdx.y;
	unsigned int bIDx = blockIdx.x;
	unsigned int bIDy = blockIdx.y;
														//[0,1,2,3]
	float tmpMulti = *(iPic+(bIDy+tIDy)*inD+bIDx+tIDx) * *(fPic+tIDy*fD+tIDx);

	atomicAdd(oPic+bIDy*oD+bIDx,tmpMulti);
	__syncthreads();
}





