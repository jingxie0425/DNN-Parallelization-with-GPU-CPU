#include "hip/hip_runtime.h"
/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "dense.h"
#include "funcs.h"

/*
 * call:
 * grids: {inputNum}
 * blocks: {nodeNum}
 */
__global__
void softmax(float* out, unsigned int nodeNum,float* sumExp){
	unsigned int inputNodeIndex = blockIdx.x;
	unsigned int nodeIndex = threadIdx.x;

	float expEach = exp(*(out+inputNodeIndex*nodeNum+nodeIndex));
	atomicAdd(sumExp+inputNodeIndex,expEach);
	__syncthreads();
	*(out+inputNodeIndex*nodeNum+nodeIndex) = expEach / *(sumExp+inputNodeIndex);
	__syncthreads();
}
