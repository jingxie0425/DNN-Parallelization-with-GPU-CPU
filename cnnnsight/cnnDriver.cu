/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include "cnn.h"
#include "cnncu.h"
#include "funcs.h"

//#define DEBUG
#define TIME

//not used yet
void cnnRun(float* inputs, float* filterWeights,float* outputs,float* bias,unsigned int fn, unsigned int fd, unsigned int inN,unsigned int inD,unsigned int oD,unsigned int picNum){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
	inferencePicPickHead<<<{inN,fn},1>>>(inputs,filterWeights,outputs,oD,fd,fn,inD);
	//add bias to outputs
	addBias<<<{inN,fn},{oD,oD}>>>(oD,fn,bias,outputs);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("cnnRun takes %.4f ms\n", elapsed);
#endif
}

void cnnRunNonHead(float* inputs, float* filterWeights,float* outputs,float* bias,unsigned int fn, unsigned int fd, unsigned int inN,unsigned int inD,unsigned int oD,unsigned int picNum){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
	inferencePicPickNonHead<<<{inN,picNum,fn},1>>>(inputs, filterWeights,outputs,oD,fd,fn,picNum,inD);
	//add bias to outputs
	addBias<<<{inN,fn},{oD,oD}>>>(oD,fn,bias,outputs);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("cnnRunNonHead takes %.4f ms\n", elapsed);
#endif
}

void CnnFree(CNNHANDLER cnnhandler){
	hipFree(cnnhandler->filterWeights);
	hipFree(cnnhandler->bias);
	hipFree(cnnhandler->inputs);
	hipFree(cnnhandler->outputs);

	free(cnnhandler);
}
