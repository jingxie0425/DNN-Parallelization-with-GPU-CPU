/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include "dataTrans.h"
#include <iostream>

using namespace std;

DataTrans::DataTrans(READHANDLER cpuHandler){
    cpuH = cpuHandler;
}

void DataTrans::Trans(){
    //data need to be put in GPU structure (more than two nested-level) when created
}


void DataTrans::TransBack(float* cpu_d3, float* gpuout,unsigned int oD,unsigned int oN){
	hipMemcpy(cpu_d3,gpuout,oN*oD*oD*sizeof(float),hipMemcpyDeviceToHost);

}

void DataTrans::TransBackDense(DENSEHANDLER densehandler, float* realOut,unsigned int inN,unsigned int nodeN){
	hipMemcpy(realOut,densehandler->outs,nodeN*inN*sizeof(float),hipMemcpyDeviceToHost);
}

