#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "cnn.h"
/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include "cnncu.h"
#include "maxpooling.h"
//call:
//grid = {oN}
//block = {oD/MAXPOOLINGSIZE,oD/MAXPOOLINGSIZE}
__global__
void maxpooling(float* outputs,unsigned int oD,unsigned int steps, float* out, unsigned int newOD){
	unsigned int outNum = blockIdx.x;
	unsigned int oDX = threadIdx.x;
	unsigned int oDY = threadIdx.y;
	//unsigned int fN = cnnhandler->filterNum;
	float *opic = outputs+(outNum)*oD*oD;
	float *nopic = out+(outNum)*newOD*newOD;
	float max = 0;
	//choose the max within step range
	for(int i = 0; i < steps; i++){
		for(int j = 0; j<steps;j++){
			if((steps*oDY+i)<oD && (steps*oDX+j)<oD){
				//zero padding
				if(*(opic+(steps*oDY+i)*oD+steps*oDX+j) > max)
									max = *(opic+(steps*oDY+i)*oD+steps*oDX+j);
			}

		}
	}
	//store max into the root position
	*(nopic+oDY*newOD+oDX) = max;
	__syncthreads();
}

//call:
//grid = {oN}
//block = {oD/MAXPOOLINGSIZE,oD/MAXPOOLINGSIZE}
__global__
void agragate(float* outputs,unsigned int oD,unsigned int steps){
	unsigned int outNum = blockIdx.x;
	unsigned int oDX = threadIdx.x;
	unsigned int oDY = threadIdx.y;
	float *opic = outputs+(outNum)*oD*oD;
	float max = *(opic+(steps*oDY)*oD+steps*oDX);
	//wait everyone get max
	__syncthreads();
	//resign max
	*(opic+(steps*oDY-oDY)*oD+steps*oDX-oDX) = max;
}

