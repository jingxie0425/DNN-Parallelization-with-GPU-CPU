/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include "funcs.h"
#include "softmax.h"

#define TIME

SOFTMAXHANDLER softmaxNew(unsigned int inN){
	SOFTMAXHANDLER softmaxhandler = (SOFTMAXHANDLER)malloc(sizeof(SOFTMAXSTATE));
	hipMalloc(&(softmaxhandler->sumExp),inN*sizeof(float));
	return softmaxhandler;
}

void softmaxRun(SOFTMAXHANDLER softmaxhandler,float* outs,unsigned int inN, unsigned int nodeNum){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif	
	//grids: {inputNum,nodeNum}
	//blocks: {1}
	softmax<<<inN,nodeNum>>>(outs, nodeNum,softmaxhandler->sumExp);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("softmaxRun takes %.4f ms\n", elapsed);
#endif
}

void softmaxFree(SOFTMAXHANDLER softmaxhandler){
	hipFree(softmaxhandler->sumExp);
	free(softmaxhandler);
}
