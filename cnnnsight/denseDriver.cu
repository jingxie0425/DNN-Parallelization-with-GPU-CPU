#include "hip/hip_runtime.h"
/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include "dense.h"
#include "densecu.h"
#include "funcs.h"

#define TIME

void denserun(DENSEHANDLER densehandler,unsigned int inN,unsigned int nodeNum,float* preOut,unsigned int oD, unsigned int fN){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
	//grid:{inN,oD*oD*fN,nodeN} here oD is new oD after max pooling
	//block:{1}
	forward<<<{inN,oD*oD,fN},nodeNum>>>(densehandler->outs, densehandler->Weights,preOut,oD,fN,nodeNum);
	//grid: {inN,nodeN}
	//block: {1}
	addBias<<<{inN,nodeNum},1>>>(densehandler->outs,densehandler->bias,nodeNum);
	//apply relu nonlinearlity to outputs
	//reluDense<<<{inN*nodeNum},1>>>(densehandler->outs);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("denserun takes %.4f ms\n", elapsed);
#endif
}


void denserunD(DENSEHANDLER densehandler,unsigned int inN,unsigned int preNodeNum, unsigned int nodeNum,float* predenseouts){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
	//grid:{inN,preNodeNum,nodeN}
	//block:{1}
	forwardD<<<{inN,preNodeNum,nodeNum},1>>>(predenseouts,densehandler->Weights,densehandler->outs, preNodeNum, nodeNum);
	//grid: {inN,nodeN}
	//block: {1}
	addBias<<<{inN,nodeNum},1>>>(densehandler->outs,densehandler->bias,nodeNum);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("denserunD takes %.4f ms\n", elapsed);
#endif
}

void headDenserunD(HEADDENSEHANDLER headdensehandler,unsigned int inN,unsigned int preNodeNum, unsigned int nodeNum,float* predenseouts){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
	//grid:{inN,preNodeNum,nodeN}
	//block:{1}
	forwardD<<<{inN,preNodeNum,nodeNum},1>>>(predenseouts,headdensehandler->Weights,headdensehandler->outs, preNodeNum, nodeNum);
	//grid: {inN,nodeN}
	//block: {1}
	addBias<<<{inN,nodeNum},1>>>(headdensehandler->outs,headdensehandler->bias,nodeNum);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("headDenserunD takes %.4f ms\n", elapsed);
#endif
}

void denseFree(DENSEHANDLER densehandler){
	hipFree(densehandler->Weights);
	hipFree(densehandler->bias);
	hipFree(densehandler->outs);
	free(densehandler);
}

void headDenseFree(HEADDENSEHANDLER headdensehandler){
	hipFree(headdensehandler->Weights);
	hipFree(headdensehandler->bias);
	hipFree(headdensehandler->outs);
	hipFree(headdensehandler->inputs);

	free(headdensehandler);
}