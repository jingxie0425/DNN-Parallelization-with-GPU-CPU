/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include <stdio.h>
#include <iostream> 
#include "funcs.h"
#include "relu.h"

#define TIME

void reluCnn( float* outputs,unsigned int fn, unsigned int inN,unsigned int oD){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif	
	relu<<<{inN*fn},{oD,oD}>>>(oD,outputs);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("reluCnn takes %.4f ms\n", elapsed);
#endif
}


void reluDen(float* outputs,unsigned int inN,unsigned int nodeNum){
#ifdef TIME
	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif	
	reluDense<<<{inN*nodeNum},1>>>(outputs);
#ifdef TIME
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("reluDen takes %.4f ms\n", elapsed);
#endif
}
