/*
 * @author: xiaomin wu
 * @date: 1/16/2020
 * */
#include "writeOut.h"

void writeTrans(float* cpudata,float* gpudata, unsigned int d1,unsigned int d2){
	hipMemcpy(cpudata,gpudata,d1*d2*sizeof(float),hipMemcpyDeviceToHost);
}

